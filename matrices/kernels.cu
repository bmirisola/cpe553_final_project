#include "hip/hip_runtime.h"
#include "kernels.cuh"


using namespace std;

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(float *A, float *B, float *C, int N){

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}

void matrixOperations(GtkWidget *widget, gpointer data) {
    //matrixMultiplication()
    gchar* b = gtk_combo_box_text_get_active_text(GTK_COMBO_BOX_TEXT(data));
    int size = 0;
    bool isFirstMatrixDone, isSecondMatrixDone = false;

    fstream file ("/home/bmirisola/CLionProjects/cpe553/cpe553_final_project/matrices/matrices.csv", ios::in);

    string line, word;
    vector<string> row;
    vector<vector<string>> matrix1;
    vector<vector<string>> matrix2;

    if(file.is_open()){
        while(getline(file,line)){
            row.clear();

            stringstream s(line);

            while(getline(s,word,',')) {
                if(word == "op"){
                    isFirstMatrixDone = true;
                }
                if(word != "=====" && word != "op" ) {
                    row.push_back(word);
                }
            }

            if(!isFirstMatrixDone){
                matrix1.push_back(row);
                size++;
            }
            else if (!row.empty()) {
                matrix2.push_back(row);
            }
        }
    }

    
    if(b == NULL) {
        g_print("Remember to pick an operation");
    }

    else if(g_content_type_equals(b,"*")) {

    }

    else if(g_content_type_equals(b,"+")) {
        g_print("Done +");
    }

    else if(g_content_type_equals(b,"-")) {
        g_print("Done -");
    }

}