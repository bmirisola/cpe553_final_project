#include "hip/hip_runtime.h"
#include "kernels.cuh"


using namespace std;

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(float *A, float *B, float *C, int N){

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}

void matrixOperations(GtkWidget *widget, gpointer data) {
    //matrixMultiplication()
    gchar* b = gtk_combo_box_text_get_active_text(GTK_COMBO_BOX_TEXT(data));
    int size = 0;
    bool isFirstMatrixDone, isSecondMatrixDone = false;

    fstream file ("/home/bmirisola/CLionProjects/cpe553/cpe553_final_project/matrices/matrices.csv", ios::in);

    string line, word;
    vector<string> matrix1;
    vector<string> matrix2;
    vector<string> ccc;

    if(file.is_open()){
        while(getline(file,line)){

            stringstream s(line);

            while(getline(s,word,',')) {
                if(word == "op"){
                    isFirstMatrixDone = true;
                }
                if(word != "=====" && word != "op" && !isFirstMatrixDone) {
                    matrix1.push_back(word);
                }
                else if(word != "=====" && word != "op" && isFirstMatrixDone){
                    matrix2.push_back(word);
                }
            }

            if(!isFirstMatrixDone){
                size++;
            }
        }
    }
    size = pow(size,2);
    for(int i = 0; i<size; i++){
        cout << matrix2[i] << " ";
    }

    dev_array<double> d_A(size*size);
    dev_array<double> d_B(size*size);
    dev_array<double> d_C(size*size);

    if(b == NULL) {
        g_print("Remember to pick an operation");
    }

    else if(g_content_type_equals(b,"*")) {

    }

    else if(g_content_type_equals(b,"+")) {
        g_print("Done +");
    }

    else if(g_content_type_equals(b,"-")) {
        g_print("Done -");
    }

}