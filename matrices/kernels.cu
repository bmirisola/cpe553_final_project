#include "hip/hip_runtime.h"
#include "kernels.cuh"


using namespace std;

__global__ void matrixMultiplicationKernel(double *A, double *B, double *C, int N) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(double *A, double *B, double *C, int N) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
}

void matrixOperations(GtkWidget *widget, gpointer data) {
    //matrixMultiplication()
    gchar *b = gtk_combo_box_text_get_active_text(GTK_COMBO_BOX_TEXT(data));
    int size = 0;
    bool isFirstMatrixDone, isSecondMatrixDone = false;

    fstream file("/home/bmirisola/CLionProjects/cpe553/cpe553_final_project/matrices/matrices.csv", ios::in);

    string line, word;
    vector<string> matrix1;
    vector<string> matrix2;
    vector<string> ccc;

    if (file.is_open()) {
        while (getline(file, line)) {

            stringstream s(line);

            while (getline(s, word, ',')) {
                if (word == "op") {
                    isFirstMatrixDone = true;
                }
                if (word != "=====" && word != "op" && !isFirstMatrixDone) {
                    matrix1.push_back(word);
                } else if (word != "=====" && word != "op" && isFirstMatrixDone) {
                    matrix2.push_back(word);
                }
            }

            if (!isFirstMatrixDone) {
                size++;
            }
        }
    }
    size = pow(size, 2);

    vector<double> h_A(size);
    vector<double> h_B(size);
    vector<double> h_C(size);

    for (int i = 0; i < size; i++){
        h_A[i] = stoi(matrix1[i]);
        h_B[i] = stoi(matrix2[i]);
    }

    dev_array<double> d_A( size);
    dev_array<double> d_B(size);
    dev_array<double> d_C(size);

    d_A.set(&h_A[0], size);
    d_B.set(&h_B[0], size);

    if (b == NULL) {
        g_print("Remember to pick an operation");
    } else if (g_content_type_equals(b, "*")) {
        matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), (int)sqrt(size));
        hipDeviceSynchronize();

        d_C.copyToHost(&h_C[0], size);
        hipDeviceSynchronize();

        for (int i = 0; i < size; i++){
            g_print("%f ", h_C[i]);
        }
    } else if (g_content_type_equals(b, "+")) {
        g_print("Done +");
    } else if (g_content_type_equals(b, "-")) {
        g_print("Done -");
    }

}