#include "hip/hip_runtime.h"
#include "kernels.cuh"

using namespace std;

__global__ void matrixMultiplicationKernel(double *A, double *B, double *C, int N) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    double tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}

__global__ void matrixAdditionKernel(double *A, double *B, double *C, int N) {
    // Grid stride loop
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // each thread computes one element of the block sub-matrix
    for (int i = index; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }


}

__global__ void matrixSubtractionKernel(double *A, double *B, double *C, int N) {

    // Grid stride loop
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // each thread computes one element of the block sub-matrix
    for (int i = index; i < N; i += stride) {
        C[i] = A[i] - B[i];
    }
}

void matrixMultiplication(double *A, double *B, double *C, int N) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
}

void matrixAddition(double *A, double *B, double *C, int N) {
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matrixAdditionKernel<<<1, threadsPerBlock>>>(A, B, C, N);
}

void matrixSubtraction(double *A, double *B, double *C, int N) {
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(256, 256);

    matrixSubtractionKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
}

void matrixOperations(GtkWidget *widget, gpointer data) {

    // Parse combobox text
    gchar *b = gtk_combo_box_text_get_active_text(GTK_COMBO_BOX_TEXT(data));
    int size = 0; //set size of matrix to 0
    bool isFirstMatrixDone = false; // boolean to check whether the first matrix has been parsed from file

    //File taht holds matrices
    fstream file("/home/bmirisola/CLionProjects/cpe553/cpe553_final_project/matrices/matrices.csv", ios::in);

    // variables to hold csv file values
    string line, word;
    vector<string> matrix1;
    vector<string> matrix2;

    /*
     * If statement parses preconstructed csv file for matrices
     * open file previously created
     * read line by line
     * convert line to string stream
     * add each number to the first matrix until string is read
     * Start adding numbers to second matrix
     */

    if (file.is_open()) {
        while (getline(file, line)) {

            stringstream s(line);

            while (getline(s, word, ',')) {
                if (word == "op") {
                    isFirstMatrixDone = true;
                }
                if (word != "=====" && word != "op" && !isFirstMatrixDone) {
                    matrix1.push_back(word);
                } else if (word != "=====" && word != "op" && isFirstMatrixDone) {
                    matrix2.push_back(word);
                }
            }

            if (!isFirstMatrixDone) {
                //Only add to size during first matrix
                size++;
            }
        }
    }

    //square size to make vector have rows and columns of size size
    size = pow(size, 2);

    //Create host matrices
    vector<double> h_A(size);
    vector<double> h_B(size);
    vector<double> h_C(size);

    //Populate host matrices with values from parsed matrices and convert to double
    for (int i = 0; i < size; i++) {
        h_A[i] = stod(matrix1[i]);
        h_B[i] = stod(matrix2[i]);
    }

    //Create dev arrays to hold vectors and manage gpu memory
    dev_array<double> d_A(size);
    dev_array<double> d_B(size);
    dev_array<double> d_C(size);

    //Set matrices to device
    d_A.set(&h_A[0], size);
    d_B.set(&h_B[0], size);

    /*
     * Checks value of b for operation
     * Launches appropriate gpu kernel
     * Writes to result array so cpu can read it
     * Synchronizes with the main thread so the program does not conclude before kernel execution
     */

    if (b == NULL) {
        g_print("Remember to pick an operation");
    } else if (g_content_type_equals(b, "*")) {
        matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), (int) sqrt(size));
        hipDeviceSynchronize();

        d_C.copyToHost(&h_C[0], size);
        hipDeviceSynchronize();
    } else if (g_content_type_equals(b, "+")) {
        matrixAddition(d_A.getData(), d_B.getData(), d_C.getData(), (int) (size));
        hipDeviceSynchronize();

        d_C.copyToHost(&h_C[0], size);
        hipDeviceSynchronize();
    } else if (g_content_type_equals(b, "-")) {
        matrixSubtraction(d_A.getData(), d_B.getData(), d_C.getData(), (int) size);
        hipDeviceSynchronize();

        d_C.copyToHost(&h_C[0], size);
        hipDeviceSynchronize();
    }

    //Close file and free recource
    file.close();

    // Write result to txt file
    if (!h_C.empty()) {
        int n = (int) sqrt(size);
        ofstream resultFile("/home/bmirisola/CLionProjects/cpe553/cpe553_final_project/matrices/result.txt");
        if (resultFile.is_open()) {
            for (int i = 0; i < n; i++) {
                for (int j = 0; j < n; j++) {
                    resultFile << h_C[i * n + j] << " ";
                }
                resultFile << "\n";
            }
        }

        //Close file and free recource
        resultFile.close();
    }
}